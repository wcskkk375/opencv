#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2008-2012, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/core/cuda_types.hpp"
#include "opencv2/core/cuda/common.hpp"

namespace cv { namespace softcascade { namespace cudev
{
    typedef unsigned int uint;
    typedef unsigned short ushort;

    // Utility function to extract unsigned chars from an unsigned integer
    __device__ uchar4 int_to_uchar4(unsigned int in)
    {
        uchar4 bytes;
        bytes.x = (in & 0x000000ff) >>  0;
        bytes.y = (in & 0x0000ff00) >>  8;
        bytes.z = (in & 0x00ff0000) >> 16;
        bytes.w = (in & 0xff000000) >> 24;
        return bytes;
    }

    __global__ void shfl_integral_horizontal(const cv::cuda::PtrStep<uint4> img, cv::cuda::PtrStep<uint4> integral)
    {
    #if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 300)
        __shared__ int sums[128];

        const int id = threadIdx.x;
        const int lane_id = id % warpSize;
        const int warp_id = id / warpSize;

        const uint4 data = img(blockIdx.x, id);

        const uchar4 a = int_to_uchar4(data.x);
        const uchar4 b = int_to_uchar4(data.y);
        const uchar4 c = int_to_uchar4(data.z);
        const uchar4 d = int_to_uchar4(data.w);

        int result[16];

        result[0]  =              a.x;
        result[1]  = result[0]  + a.y;
        result[2]  = result[1]  + a.z;
        result[3]  = result[2]  + a.w;

        result[4]  = result[3]  + b.x;
        result[5]  = result[4]  + b.y;
        result[6]  = result[5]  + b.z;
        result[7]  = result[6]  + b.w;

        result[8]  = result[7]  + c.x;
        result[9]  = result[8]  + c.y;
        result[10] = result[9]  + c.z;
        result[11] = result[10] + c.w;

        result[12] = result[11] + d.x;
        result[13] = result[12] + d.y;
        result[14] = result[13] + d.z;
        result[15] = result[14] + d.w;

        int sum = result[15];

        // the prefix sum for each thread's 16 value is computed,
        // now the final sums (result[15]) need to be shared
        // with the other threads and add.  To do this,
        // the __shfl_up() instruction is used and a shuffle scan
        // operation is performed to distribute the sums to the correct
        // threads
        #pragma unroll
        for (int i = 1; i < 32; i *= 2)
        {
            const int n = __shfl_up(sum, i, 32);

            if (lane_id >= i)
            {
                #pragma unroll
                for (int i = 0; i < 16; ++i)
                    result[i] += n;

                sum += n;
            }
        }

        // Now the final sum for the warp must be shared
        // between warps.  This is done by each warp
        // having a thread store to shared memory, then
        // having some other warp load the values and
        // compute a prefix sum, again by using __shfl_up.
        // The results are uniformly added back to the warps.
        // last thread in the warp holding sum of the warp
        // places that in shared
        if (threadIdx.x % warpSize == warpSize - 1)
            sums[warp_id] = result[15];

        __syncthreads();

        if (warp_id == 0)
        {
            int warp_sum = sums[lane_id];

            #pragma unroll
            for (int i = 1; i <= 32; i *= 2)
            {
                const int n = __shfl_up(warp_sum, i, 32);

                if (lane_id >= i)
                    warp_sum += n;
            }

            sums[lane_id] = warp_sum;
        }

        __syncthreads();

        int blockSum = 0;

        // fold in unused warp
        if (warp_id > 0)
        {
            blockSum = sums[warp_id - 1];

            #pragma unroll
            for (int i = 0; i < 16; ++i)
                result[i] += blockSum;
        }

        // assemble result
        // Each thread has 16 values to write, which are
        // now integer data (to avoid overflow).  Instead of
        // each thread writing consecutive uint4s, the
        // approach shown here experiments using
        // the shuffle command to reformat the data
        // inside the registers so that each thread holds
        // consecutive data to be written so larger contiguous
        // segments can be assembled for writing.

        /*
            For example data that needs to be written as

            GMEM[16] <- x0 x1 x2 x3 y0 y1 y2 y3 z0 z1 z2 z3 w0 w1 w2 w3
            but is stored in registers (r0..r3), in four threads (0..3) as:

            threadId   0  1  2  3
              r0      x0 y0 z0 w0
              r1      x1 y1 z1 w1
              r2      x2 y2 z2 w2
              r3      x3 y3 z3 w3

              after apply __shfl_xor operations to move data between registers r1..r3:

            threadId  00 01 10 11
                      x0 y0 z0 w0
             xor(01)->y1 x1 w1 z1
             xor(10)->z2 w2 x2 y2
             xor(11)->w3 z3 y3 x3

             and now x0..x3, and z0..z3 can be written out in order by all threads.

             In the current code, each register above is actually representing
             four integers to be written as uint4's to GMEM.
        */

        result[4]  = __shfl_xor(result[4] , 1, 32);
        result[5]  = __shfl_xor(result[5] , 1, 32);
        result[6]  = __shfl_xor(result[6] , 1, 32);
        result[7]  = __shfl_xor(result[7] , 1, 32);

        result[8]  = __shfl_xor(result[8] , 2, 32);
        result[9]  = __shfl_xor(result[9] , 2, 32);
        result[10] = __shfl_xor(result[10], 2, 32);
        result[11] = __shfl_xor(result[11], 2, 32);

        result[12] = __shfl_xor(result[12], 3, 32);
        result[13] = __shfl_xor(result[13], 3, 32);
        result[14] = __shfl_xor(result[14], 3, 32);
        result[15] = __shfl_xor(result[15], 3, 32);

        uint4* integral_row = integral.ptr(blockIdx.x);
        uint4 output;

        ///////

        if (threadIdx.x % 4 == 0)
            output = make_uint4(result[0], result[1], result[2], result[3]);

        if (threadIdx.x % 4 == 1)
            output = make_uint4(result[4], result[5], result[6], result[7]);

        if (threadIdx.x % 4 == 2)
            output = make_uint4(result[8], result[9], result[10], result[11]);

        if (threadIdx.x % 4 == 3)
            output = make_uint4(result[12], result[13], result[14], result[15]);

        integral_row[threadIdx.x % 4 + (threadIdx.x / 4) * 16] = output;

        ///////

        if (threadIdx.x % 4 == 2)
            output = make_uint4(result[0], result[1], result[2], result[3]);

        if (threadIdx.x % 4 == 3)
            output = make_uint4(result[4], result[5], result[6], result[7]);

        if (threadIdx.x % 4 == 0)
            output = make_uint4(result[8], result[9], result[10], result[11]);

        if (threadIdx.x % 4 == 1)
            output = make_uint4(result[12], result[13], result[14], result[15]);

        integral_row[(threadIdx.x + 2) % 4 + (threadIdx.x / 4) * 16 + 8] = output;

        // continuning from the above example,
        // this use of __shfl_xor() places the y0..y3 and w0..w3 data
        // in order.

        #pragma unroll
        for (int i = 0; i < 16; ++i)
            result[i] = __shfl_xor(result[i], 1, 32);

        if (threadIdx.x % 4 == 0)
            output = make_uint4(result[0], result[1], result[2], result[3]);

        if (threadIdx.x % 4 == 1)
            output = make_uint4(result[4], result[5], result[6], result[7]);

        if (threadIdx.x % 4 == 2)
            output = make_uint4(result[8], result[9], result[10], result[11]);

        if (threadIdx.x % 4 == 3)
            output = make_uint4(result[12], result[13], result[14], result[15]);

        integral_row[threadIdx.x % 4 + (threadIdx.x / 4) * 16 + 4] = output;

        ///////

        if (threadIdx.x % 4 == 2)
            output = make_uint4(result[0], result[1], result[2], result[3]);

        if (threadIdx.x % 4 == 3)
            output = make_uint4(result[4], result[5], result[6], result[7]);

        if (threadIdx.x % 4 == 0)
            output = make_uint4(result[8], result[9], result[10], result[11]);

        if (threadIdx.x % 4 == 1)
            output = make_uint4(result[12], result[13], result[14], result[15]);

        integral_row[(threadIdx.x + 2) % 4 + (threadIdx.x / 4) * 16 + 12] = output;
    #endif
    }

    // This kernel computes columnwise prefix sums.  When the data input is
    // the row sums from above, this completes the integral image.
    // The approach here is to have each block compute a local set of sums.
    // First , the data covered by the block is loaded into shared memory,
    // then instead of performing a sum in shared memory using __syncthreads
    // between stages, the data is reformatted so that the necessary sums
    // occur inside warps and the shuffle scan operation is used.
    // The final set of sums from the block is then propgated, with the block
    // computing "down" the image and adding the running sum to the local
    // block sums.
    __global__ void shfl_integral_vertical(cv::cuda::PtrStepSz<unsigned int> integral)
    {
    #if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 300)
        __shared__ unsigned int sums[32][9];

        const int tidx = blockIdx.x * blockDim.x + threadIdx.x;
        const int lane_id = tidx % 8;

        if (tidx >= integral.cols)
            return;

        sums[threadIdx.x][threadIdx.y] = 0;
        __syncthreads();

        unsigned int stepSum = 0;

        for (int y = threadIdx.y; y < integral.rows; y += blockDim.y)
        {
            unsigned int* p = integral.ptr(y) + tidx;

            unsigned int sum = *p;

            sums[threadIdx.x][threadIdx.y] = sum;
            __syncthreads();

            // place into SMEM
            // shfl scan reduce the SMEM, reformating so the column
            // sums are computed in a warp
            // then read out properly
            const int j = threadIdx.x % 8;
            const int k = threadIdx.x / 8 + threadIdx.y * 4;

            int partial_sum = sums[k][j];

            for (int i = 1; i <= 8; i *= 2)
            {
                int n = __shfl_up(partial_sum, i, 32);

                if (lane_id >= i)
                    partial_sum += n;
            }

            sums[k][j] = partial_sum;
            __syncthreads();

            if (threadIdx.y > 0)
                sum += sums[threadIdx.x][threadIdx.y - 1];

            sum += stepSum;
            stepSum += sums[threadIdx.x][blockDim.y - 1];

            __syncthreads();

            *p = sum;
        }
    #endif
    }

    void shfl_integral(const cv::cuda::PtrStepSzb& img, cv::cuda::PtrStepSz<unsigned int> integral, hipStream_t stream)
    {
        {
            // each thread handles 16 values, use 1 block/row
            // save, becouse step is actually can't be less 512 bytes
            int block = integral.cols / 16;

            // launch 1 block / row
            const int grid = img.rows;

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(shfl_integral_horizontal), hipFuncCachePreferL1) );

            shfl_integral_horizontal<<<grid, block, 0, stream>>>((const cv::cuda::PtrStepSz<uint4>) img, (cv::cuda::PtrStepSz<uint4>) integral);
            cudaSafeCall( hipGetLastError() );
        }

        {
            const dim3 block(32, 8);
            const dim3 grid(cv::cuda::device::divUp(integral.cols, block.x), 1);

            shfl_integral_vertical<<<grid, block, 0, stream>>>(integral);
            cudaSafeCall( hipGetLastError() );
        }

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    __global__ void shfl_integral_vertical(cv::cuda::PtrStepSz<unsigned int> buffer, cv::cuda::PtrStepSz<unsigned int> integral)
    {
    #if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 300)
        __shared__ unsigned int sums[32][9];

        const int tidx = blockIdx.x * blockDim.x + threadIdx.x;
        const int lane_id = tidx % 8;

        if (tidx >= integral.cols)
            return;

        sums[threadIdx.x][threadIdx.y] = 0;
        __syncthreads();

        unsigned int stepSum = 0;

        for (int y = threadIdx.y; y < integral.rows; y += blockDim.y)
        {
            unsigned int* p = buffer.ptr(y) + tidx;
            unsigned int* dst = integral.ptr(y + 1) + tidx + 1;

            unsigned int sum = *p;

            sums[threadIdx.x][threadIdx.y] = sum;
            __syncthreads();

            // place into SMEM
            // shfl scan reduce the SMEM, reformating so the column
            // sums are computed in a warp
            // then read out properly
            const int j = threadIdx.x % 8;
            const int k = threadIdx.x / 8 + threadIdx.y * 4;

            int partial_sum = sums[k][j];

            for (int i = 1; i <= 8; i *= 2)
            {
                int n = __shfl_up(partial_sum, i, 32);

                if (lane_id >= i)
                    partial_sum += n;
            }

            sums[k][j] = partial_sum;
            __syncthreads();

            if (threadIdx.y > 0)
                sum += sums[threadIdx.x][threadIdx.y - 1];

            sum += stepSum;
            stepSum += sums[threadIdx.x][blockDim.y - 1];

            __syncthreads();

            *dst = sum;
        }
    #endif
    }

    // used for frame preprocessing before Soft Cascade evaluation: no synchronization needed
    void shfl_integral_gpu_buffered(cv::cuda::PtrStepSzb img, cv::cuda::PtrStepSz<uint4> buffer, cv::cuda::PtrStepSz<unsigned int> integral,
        int blockStep, hipStream_t stream)
    {
        {
            const int block = blockStep;
            const int grid = img.rows;

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(shfl_integral_horizontal), hipFuncCachePreferL1) );

            shfl_integral_horizontal<<<grid, block, 0, stream>>>((cv::cuda::PtrStepSz<uint4>) img, buffer);
            cudaSafeCall( hipGetLastError() );
        }

        {
            const dim3 block(32, 8);
            const dim3 grid(cv::cuda::device::divUp(integral.cols, block.x), 1);

            shfl_integral_vertical<<<grid, block, 0, stream>>>((cv::cuda::PtrStepSz<unsigned int>)buffer, integral);
            cudaSafeCall( hipGetLastError() );
        }
    }
    // 0
#define CV_DESCALE(x, n) (((x) + (1 << ((n)-1))) >> (n))

    enum
    {
        yuv_shift  = 14,
        xyz_shift  = 12,
        R2Y        = 4899,
        G2Y        = 9617,
        B2Y        = 1868
    };

    template <int bidx> static __device__ __forceinline__ unsigned char RGB2GrayConvert(unsigned char b, unsigned char g, unsigned char r)
    {
        // uint b = 0xffu & (src >> (bidx * 8));
        // uint g = 0xffu & (src >> 8);
        // uint r = 0xffu & (src >> ((bidx ^ 2) * 8));
        return CV_DESCALE((unsigned int)(b * B2Y + g * G2Y + r * R2Y), yuv_shift);
    }

    __global__ void device_transform(const cv::cuda::PtrStepSz<uchar3> bgr, cv::cuda::PtrStepSzb gray)
    {
        const int y = blockIdx.y * blockDim.y + threadIdx.y;
        const int x = blockIdx.x * blockDim.x + threadIdx.x;

        const uchar3 colored = (uchar3)(bgr.ptr(y))[x];

        gray.ptr(y)[x] = RGB2GrayConvert<0>(colored.x, colored.y, colored.z);
    }

    ///////
    void transform(const cv::cuda::PtrStepSz<uchar3>& bgr, cv::cuda::PtrStepSzb gray)
    {
        const dim3 block(32, 8);
        const dim3 grid(cv::cuda::device::divUp(bgr.cols, block.x), cv::cuda::device::divUp(bgr.rows, block.y));
        device_transform<<<grid, block>>>(bgr, gray);
        cudaSafeCall(hipDeviceSynchronize());
    }
}}}
