#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2008-2012, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include <cuda_invoker.hpp>
#include <float.h>
#include <stdio.h>
#include "opencv2/core/cuda/common.hpp"

namespace cv { namespace softcascade { namespace cudev {

typedef unsigned char uchar;

    template <int FACTOR>
    __device__ __forceinline__ uchar shrink(const uchar* ptr, const int pitch, const int y, const int x)
    {
        int out = 0;
#pragma unroll
        for(int dy = 0; dy < FACTOR; ++dy)
#pragma unroll
            for(int dx = 0; dx < FACTOR; ++dx)
            {
                out += ptr[dy * pitch + dx];
            }

        return static_cast<uchar>(out / (FACTOR * FACTOR));
    }

    template<int FACTOR>
    __global__ void shrink(const uchar* __restrict__ hogluv, const size_t inPitch,
                                 uchar* __restrict__ shrank, const size_t outPitch )
    {
        const int y = blockIdx.y * blockDim.y + threadIdx.y;
        const int x = blockIdx.x * blockDim.x + threadIdx.x;

        const uchar* ptr = hogluv + (FACTOR * y) * inPitch + (FACTOR * x);

        shrank[ y * outPitch + x] = shrink<FACTOR>(ptr, inPitch, y, x);
    }

    void shrink(const cv::cuda::PtrStepSzb& channels, cv::cuda::PtrStepSzb shrunk)
    {
        dim3 block(32, 8);
        dim3 grid(shrunk.cols / 32, shrunk.rows / 8);
        shrink<4><<<grid, block>>>((uchar*)channels.ptr(), channels.step, (uchar*)shrunk.ptr(), shrunk.step);
        cudaSafeCall(hipDeviceSynchronize());
    }

    __device__ __forceinline__ void luv(const float& b, const float& g, const float& r, uchar& __l, uchar& __u, uchar& __v)
    {
        // rgb -> XYZ
        float x = 0.412453f * r + 0.357580f * g + 0.180423f * b;
        float y = 0.212671f * r + 0.715160f * g + 0.072169f * b;
        float z = 0.019334f * r + 0.119193f * g + 0.950227f * b;

        // computed for D65
        const float _ur = 0.19783303699678276f;
        const float _vr = 0.46833047435252234f;

        const float divisor = fmax((x + 15.f * y + 3.f * z), FLT_EPSILON);
        const float _u = __fdividef(4.f * x, divisor);
        const float _v = __fdividef(9.f * y, divisor);

        float hack = static_cast<float>(__float2int_rn(y * 2047)) / 2047;
        const float L = fmax(0.f, ((116.f * cbrtf(hack)) - 16.f));
        const float U = 13.f * L * (_u - _ur);
        const float V = 13.f * L * (_v - _vr);

        // L in [0, 100], u in [-134, 220], v in [-140, 122]
        __l = static_cast<uchar>( L * (255.f / 100.f));
        __u = static_cast<uchar>((U + 134.f) * (255.f / (220.f + 134.f )));
        __v = static_cast<uchar>((V + 140.f) * (255.f / (122.f + 140.f )));
    }

    __global__ void bgr2Luv_d(const uchar* rgb, const size_t rgbPitch, uchar* luvg, const size_t luvgPitch)
    {
        const int y = blockIdx.y * blockDim.y + threadIdx.y;
        const int x = blockIdx.x * blockDim.x + threadIdx.x;

        uchar3 color = ((uchar3*)(rgb + rgbPitch * y))[x];
        uchar l, u, v;
        luv(color.x / 255.f, color.y / 255.f, color.z / 255.f, l, u, v);

        luvg[luvgPitch *  y + x] = l;
        luvg[luvgPitch * (y + 480) + x] = u;
        luvg[luvgPitch * (y + 2 * 480) + x] = v;
    }

    void bgr2Luv(const cv::cuda::PtrStepSzb& bgr, cv::cuda::PtrStepSzb luv)
    {
        dim3 block(32, 8);
        dim3 grid(bgr.cols / 32, bgr.rows / 8);

        bgr2Luv_d<<<grid, block>>>((const uchar*)bgr.ptr(0), bgr.step, (uchar*)luv.ptr(0), luv.step);

        cudaSafeCall(hipDeviceSynchronize());
    }

    template<bool isDefaultNum>
    __device__ __forceinline__ int fast_angle_bin(const float& dx, const float& dy)
    {
        const float angle_quantum = CV_PI_F / 6.f;
        float angle = atan2(dx, dy) + (angle_quantum / 2.f);

        if (angle < 0) angle += CV_PI_F;

        const float angle_scaling = 1.f / angle_quantum;
        return static_cast<int>(angle * angle_scaling) % 6;
    }

    template<>
    __device__ __forceinline__ int fast_angle_bin<true>(const float& dy, const float& dx)
    {
        int index = 0;

        float max_dot = fabs(dx);

        {
            const float dot_product = fabs(dx * 0.8660254037844386f + dy * 0.5f);

            if(dot_product > max_dot)
            {
                max_dot = dot_product;
                index = 1;
            }
        }
        {
            const float dot_product = fabs(dy * 0.8660254037844386f + dx * 0.5f);

            if(dot_product > max_dot)
            {
                max_dot = dot_product;
                index = 2;
            }
        }
        {
            int i = 3;
            float2 bin_vector_i;
            bin_vector_i.x = ::cos(i * (CV_PI_F / 6.f));
            bin_vector_i.y = ::sin(i * (CV_PI_F / 6.f));

            const float dot_product = fabs(dx * bin_vector_i.x + dy * bin_vector_i.y);
            if(dot_product > max_dot)
            {
                max_dot = dot_product;
                index = i;
            }
        }
        {
            const float dot_product = fabs(dx * (-0.4999999999999998f) + dy * 0.8660254037844387f);
            if(dot_product > max_dot)
            {
                max_dot = dot_product;
                index = 4;
            }
        }
        {
            const float dot_product = fabs(dx * (-0.8660254037844387f) + dy * 0.49999999999999994f);
            if(dot_product > max_dot)
            {
                max_dot = dot_product;
                index = 5;
            }
        }
        return index;
    }

    texture<uchar,  hipTextureType2D, hipReadModeElementType> tgray;

    template<bool isDefaultNum>
    __global__ void gray2hog(cv::cuda::PtrStepSzb mag)
    {
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        const float dx = tex2D(tgray, x + 1, y + 0) - tex2D(tgray, x - 1, y - 0);
        const float dy = tex2D(tgray, x + 0, y + 1) - tex2D(tgray, x - 0, y - 1);

        const float magnitude = sqrtf((dx * dx) + (dy * dy)) * (1.0f / sqrtf(2));
        const uchar cmag = static_cast<uchar>(magnitude);

        mag( 480 * 6 + y, x) = cmag;
        mag( 480 * fast_angle_bin<isDefaultNum>(dy, dx) + y, x) = cmag;
    }

    void gray2hog(const cv::cuda::PtrStepSzb& gray, cv::cuda::PtrStepSzb mag, const int bins)
    {
        dim3 block(32, 8);
        dim3 grid(gray.cols / 32, gray.rows / 8);

        hipChannelFormatDesc desc = hipCreateChannelDesc<uchar>();
        cudaSafeCall( hipBindTexture2D(0, tgray, gray.data, desc, gray.cols, gray.rows, gray.step) );

        if (bins == 6)
            gray2hog<true><<<grid, block>>>(mag);
        else
            gray2hog<false><<<grid, block>>>(mag);

        cudaSafeCall(hipDeviceSynchronize());
    }

    // ToDo: use textures or uncached load instruction.
    __global__ void magToHist(const uchar* __restrict__ mag,
                              const float* __restrict__ angle, const size_t angPitch,
                                    uchar* __restrict__ hog,   const size_t hogPitch, const int fh)
    {
        const int y = blockIdx.y * blockDim.y + threadIdx.y;
        const int x = blockIdx.x * blockDim.x + threadIdx.x;

        const int bin = (int)(angle[y * angPitch + x]);
        const uchar val = mag[y * hogPitch + x];
        hog[((fh * bin) + y) * hogPitch + x] = val;
    }

    void fillBins(cv::cuda::PtrStepSzb hogluv, const cv::cuda::PtrStepSzf& nangle,
                  const int fw,  const int fh, const int bins, hipStream_t stream )
    {
        const uchar* mag = (const uchar*)hogluv.ptr(fh * bins);
        uchar* hog = (uchar*)hogluv.ptr();
        const float* angle = (const float*)nangle.ptr();

        dim3 block(32, 8);
        dim3 grid(fw / 32, fh / 8);

        magToHist<<<grid, block, 0, stream>>>(mag, angle, nangle.step / sizeof(float), hog, hogluv.step, fh);
        if (!stream)
        {
            cudaSafeCall( hipGetLastError() );
            cudaSafeCall( hipDeviceSynchronize() );
        }
    }

    __device__ __forceinline__ float overlapArea(const Detection &a, const Detection &b)
    {
        int w = ::min(a.x + a.w, b.x + b.w) - ::max(a.x, b.x);
        int h = ::min(a.y + a.h, b.y + b.h) - ::max(a.y, b.y);

        return (w < 0 || h < 0)? 0.f : (float)(w * h);
    }

    texture<uint4,  hipTextureType2D, hipReadModeElementType> tdetections;

    __global__ void overlap(const uint* n, uchar* overlaps)
    {
        const int idx = threadIdx.x;
        const int total = *n;

        for (int i = idx + 1; i < total; i += 192)
        {
            const uint4 _a = tex2D(tdetections, i, 0);
            const Detection& a = *((Detection*)(&_a));
            bool excluded = false;

            for (int j = i + 1; j < total; ++j)
            {
                const uint4 _b = tex2D(tdetections, j, 0);
                const Detection& b = *((Detection*)(&_b));
                float ovl = overlapArea(a, b) / ::min(a.w * a.h, b.w * b.h);

                if (ovl > 0.65f)
                {
                    int suppessed = (a.confidence > b.confidence)? j : i;
                    overlaps[suppessed] = 1;
                    excluded = excluded || (suppessed == i);
                }

            #if defined __CUDA_ARCH__ && (__CUDA_ARCH__ >= 120)
                if (__all(excluded)) break;
            #endif
            }
        }
    }

    __global__ void collect(const uint* n, uchar* overlaps, uint* ctr, uint4* suppressed)
    {
        const int idx = threadIdx.x;
        const int total = *n;

        for (int i = idx; i < total; i += 192)
        {
            if (!overlaps[i])
            {
                int oidx = atomicInc(ctr, 50);
                suppressed[oidx] = tex2D(tdetections, i + 1, 0);
            }
        }
    }

    void suppress(const cv::cuda::PtrStepSzb& objects, cv::cuda::PtrStepSzb overlaps, cv::cuda::PtrStepSzi ndetections,
        cv::cuda::PtrStepSzb suppressed, hipStream_t stream)
    {
        int block = 192;
        int grid = 1;

        hipChannelFormatDesc desc = hipCreateChannelDesc<uint4>();
        size_t offset;
        cudaSafeCall( hipBindTexture2D(&offset, tdetections, objects.data, desc, objects.cols / sizeof(uint4), objects.rows, objects.step));

        overlap<<<grid, block>>>((uint*)ndetections.ptr(0), (uchar*)overlaps.ptr(0));
        collect<<<grid, block>>>((uint*)ndetections.ptr(0), (uchar*)overlaps.ptr(0), (uint*)suppressed.ptr(0), ((uint4*)suppressed.ptr(0)) + 1);

        if (!stream)
        {
            cudaSafeCall( hipGetLastError());
            cudaSafeCall( hipDeviceSynchronize());
        }
    }

    template<typename Policy>
    struct PrefixSum
    {
    __device_inline__ static void apply(float& impact)
        {
    #if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 300
    #pragma unroll
            // scan on shuffle functions
            for (int i = 1; i < Policy::WARP; i *= 2)
            {
                const float n = __shfl_up(impact, i, Policy::WARP);

                if (threadIdx.x >= i)
                    impact += n;
            }
    #else
            __shared__ volatile float ptr[Policy::STA_X * Policy::STA_Y];

            const int idx = threadIdx.y * Policy::STA_X + threadIdx.x;

            ptr[idx] = impact;

            if ( threadIdx.x >=  1) ptr [idx ] = (ptr [idx -  1] + ptr [idx]);
            if ( threadIdx.x >=  2) ptr [idx ] = (ptr [idx -  2] + ptr [idx]);
            if ( threadIdx.x >=  4) ptr [idx ] = (ptr [idx -  4] + ptr [idx]);
            if ( threadIdx.x >=  8) ptr [idx ] = (ptr [idx -  8] + ptr [idx]);
            if ( threadIdx.x >= 16) ptr [idx ] = (ptr [idx - 16] + ptr [idx]);

            impact = ptr[idx];
    #endif
        }
    };

    texture<int,  hipTextureType2D, hipReadModeElementType> thogluv;

    template<bool isUp>
    __device__ __forceinline__ float rescale(const Level& level, Node& node)
    {
        uchar4& scaledRect = node.rect;
        float relScale = level.relScale;
        float farea = (scaledRect.z - scaledRect.x) * (scaledRect.w - scaledRect.y);

        // rescale
        scaledRect.x = __float2int_rn(relScale * scaledRect.x);
        scaledRect.y = __float2int_rn(relScale * scaledRect.y);
        scaledRect.z = __float2int_rn(relScale * scaledRect.z);
        scaledRect.w = __float2int_rn(relScale * scaledRect.w);

        float sarea = (scaledRect.z - scaledRect.x) * (scaledRect.w - scaledRect.y);

        const float expected_new_area = farea * relScale * relScale;
        float approx = (sarea == 0)? 1: __fdividef(sarea, expected_new_area);

        float rootThreshold = (node.threshold & 0x0FFFFFFFU) * approx * level.scaling[(node.threshold >> 28) > 6];

        return rootThreshold;
    }

    template<>
    __device__ __forceinline__ float rescale<true>(const Level& level, Node& node)
    {
        uchar4& scaledRect = node.rect;
        float relScale = level.relScale;
        float farea = scaledRect.z * scaledRect.w;

        // rescale
        scaledRect.x = __float2int_rn(relScale * scaledRect.x);
        scaledRect.y = __float2int_rn(relScale * scaledRect.y);
        scaledRect.z = __float2int_rn(relScale * scaledRect.z);
        scaledRect.w = __float2int_rn(relScale * scaledRect.w);

        float sarea = scaledRect.z * scaledRect.w;

        const float expected_new_area = farea * relScale * relScale;
        float approx = __fdividef(sarea, expected_new_area);

        float rootThreshold = (node.threshold & 0x0FFFFFFFU) * approx * level.scaling[(node.threshold >> 28) > 6];

        return rootThreshold;
    }

    template<bool isUp>
    __device__ __forceinline__ int get(int x, int y, uchar4 area)
    {
        int a = tex2D(thogluv, x + area.x, y + area.y);
        int b = tex2D(thogluv, x + area.z, y + area.y);
        int c = tex2D(thogluv, x + area.z, y + area.w);
        int d = tex2D(thogluv, x + area.x, y + area.w);

        return (a - b + c - d);
    }

    template<>
    __device__ __forceinline__ int get<true>(int x, int y, uchar4 area)
    {
        x += area.x;
        y += area.y;

        int a = tex2D(thogluv, x, y);
        int b = tex2D(thogluv, x + area.z, y);
        int c = tex2D(thogluv, x + area.z, y + area.w);
        int d = tex2D(thogluv, x, y + area.w);

        return (a - b + c - d);
    }

    texture<float2,  hipTextureType2D, hipReadModeElementType> troi;

template<typename Policy>
template<bool isUp>
__device_inline__ void CascadeInvoker<Policy>::detect(Detection* objects, const uint ndetections, uint* ctr, const int downscales) const
{
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int x = blockIdx.x;

    // load Level
    __shared__ Level level;

    // check POI
    __shared__ volatile char roiCache[Policy::STA_Y];

    if (!threadIdx.y && !threadIdx.x)
        ((float2*)roiCache)[threadIdx.x] = tex2D(troi, blockIdx.y, x);

    __syncthreads();

    if (!roiCache[threadIdx.y]) return;

    if (!threadIdx.x)
        level = levels[downscales + blockIdx.z];

    if(x >= level.workRect.x || y >= level.workRect.y) return;

    int st = level.octave * level.step;
    const int stEnd = st + level.step;

    const int hogluvStep = gridDim.y * Policy::STA_Y;
    float confidence = 0.f;
    for(; st < stEnd; st += Policy::WARP)
    {
        const int nId = (st + threadIdx.x) * 3;

        Node node = nodes[nId];

        float threshold = rescale<isUp>(level, node);
        int sum = get<isUp>(x, y + (node.threshold >> 28) * hogluvStep, node.rect);

        int next = 1 + (int)(sum >= threshold);

        node = nodes[nId + next];
        threshold = rescale<isUp>(level, node);
        sum = get<isUp>(x, y + (node.threshold >> 28) * hogluvStep, node.rect);

        const int lShift = (next - 1) * 2 + (int)(sum >= threshold);
        float impact = leaves[(st + threadIdx.x) * 4 + lShift];

        PrefixSum<Policy>::apply(impact);

    #if __CUDA_ARCH__ >= 120
        if(__any((confidence + impact <= stages[(st + threadIdx.x)]))) st += 2048;
    #endif
    #if __CUDA_ARCH__ >= 300
        impact = __shfl(impact, 31);
    #endif

        confidence += impact;
    }

    if(!threadIdx.x && st == stEnd &&  ((confidence - FLT_EPSILON) >= 0))
    {
        int idx = atomicInc(ctr, ndetections);
        objects[idx] = Detection(__float2int_rn(x * Policy::SHRINKAGE),
            __float2int_rn(y * Policy::SHRINKAGE), level.objSize.x, level.objSize.y, confidence);
    }
}

template<typename Policy, bool isUp>
__global__ void soft_cascade(const CascadeInvoker<Policy> invoker, Detection* objects, const uint n, uint* ctr, const int downs)
{
    invoker.template detect<isUp>(objects, n, ctr, downs);
}

template<typename Policy>
void CascadeInvoker<Policy>::operator()(const cv::cuda::PtrStepSzb& roi, const cv::cuda::PtrStepSzi& hogluv,
    cv::cuda::PtrStepSz<uchar4> objects, const int downscales, const hipStream_t& stream) const
{
    int fw = roi.rows;
    int fh = roi.cols;

    dim3 grid(fw, fh / Policy::STA_Y, downscales);

    uint* ctr = (uint*)(objects.ptr(0));
    Detection* det = ((Detection*)objects.ptr(0)) + 1;
    uint max_det = objects.cols / sizeof(Detection);

    hipChannelFormatDesc desc = hipCreateChannelDesc<int>();
    cudaSafeCall( hipBindTexture2D(0, thogluv, hogluv.data, desc, hogluv.cols, hogluv.rows, hogluv.step));

    hipChannelFormatDesc desc_roi = hipCreateChannelDesc<typename Policy::roi_type>();
    cudaSafeCall( hipBindTexture2D(0, troi, roi.data, desc_roi, roi.cols / Policy::STA_Y, roi.rows, roi.step));

    const CascadeInvoker<Policy> inv = *this;

    soft_cascade<Policy, false><<<grid, Policy::block(), 0, stream>>>(inv, det, max_det, ctr, 0);
    cudaSafeCall( hipGetLastError());

    grid = dim3(fw, fh / Policy::STA_Y, min(38, scales) - downscales);
    soft_cascade<Policy, true><<<grid, Policy::block(), 0, stream>>>(inv, det, max_det, ctr, downscales);

    if (!stream)
    {
        cudaSafeCall( hipGetLastError());
        cudaSafeCall( hipDeviceSynchronize());
    }
}

template void CascadeInvoker<GK107PolicyX4>::operator()(const cv::cuda::PtrStepSzb& roi, const cv::cuda::PtrStepSzi& hogluv,
    cv::cuda::PtrStepSz<uchar4> objects, const int downscales, const hipStream_t& stream) const;

}}}
